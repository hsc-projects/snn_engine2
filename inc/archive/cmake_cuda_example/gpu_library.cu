#include <cuda_opengl_interop.cuh>

void Pet::register_buffer(uint b){
    id = b;
    hipGraphicsGLRegisterBuffer(&buffer_pt, id, hipGraphicsRegisterFlagsNone);
    bmapped = true;
}