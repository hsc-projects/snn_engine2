#include "hip/hip_runtime.h"
#include <construction/snn_representation.cuh>


SnnRepresentation::SnnRepresentation(
    const int N_,
    const int G_,
    const int S_,
    const int D_,

	hiprandState* rand_states_,

	float* N_pos_,
	int* G_group_delay_counts_,
    int* G_flags_, 
    float* G_props_, 
    int* N_rep_, 
    int* N_rep_buffer_, 
    int* N_rep_pre_synaptic_, 
    int* N_rep_pre_synaptic_idcs_, 
    int* N_rep_pre_synaptic_counts_, 
    int* N_delays_, 

    int* N_flags_, 
	float* N_weights_,
    
    int* L_winner_take_all_map_,
    int max_n_winner_take_all_layers_,
    int max_winner_take_all_layer_size_
){
    
	N = N_;
	G = G_;
	S = S_;
	D = D_;

	rand_states = rand_states_;

	N_pos = N_pos_;
	G_group_delay_counts = G_group_delay_counts_;
    G_flags = G_flags_; 
    G_props = G_props_; 
    N_rep = N_rep_;

	N_rep_buffer = N_rep_buffer_;
    N_rep_pre_synaptic = N_rep_pre_synaptic_; 
    N_rep_pre_synaptic_idcs = N_rep_pre_synaptic_idcs_; 
    N_rep_pre_synaptic_counts = N_rep_pre_synaptic_counts_;

    N_delays = N_delays_;

	N_flags = N_flags_;
	
	N_weights = N_weights_;

	L_winner_take_all_map = L_winner_take_all_map_;

    max_n_winner_take_all_layers = max_n_winner_take_all_layers_;
    max_winner_take_all_layer_size = max_winner_take_all_layer_size_;
}



__device__ void roll_copy(
	
	int* write_array, int* read_array, 
	int write_col, int read_col, 
	int write_row_start, 
	int n_write_array_cols, int n_read_array_cols,
	const int copy_length, 
	const int read_offset, 
	bool bprint){
	
	int write_idx;
	int read_idx;
	//int roll_mod = abs(swap_snk_N_s_start - swap_src_N_s_start);

	for (int s=0; s < abs(copy_length); s++){

		write_idx = write_col + (write_row_start + s) * n_write_array_cols;
		read_idx = read_col + (write_row_start + ((s +  read_offset) % copy_length)) * n_read_array_cols;

		if (bprint){
			printf("\nwrite_array[%d, %d]=%d -> read_array[%d, %d]=%d", 
			write_row_start + s, write_col, 
			write_array[write_idx], 
			write_row_start + ((s +  read_offset) % copy_length), read_col,
			read_array[read_idx] );
		}

		write_array[write_idx] = read_array[read_idx];

	}	

}


__device__ void shift_values_row_wise_(
	int shift_start_offset,
	int* array0, int* array1,
	int col0, int col1,
	int n_cols0, int n_cols1,
	int end_row,
	int swap_dir,
	bool bprint
){
	int idx_end0 = col0 + (end_row) * n_cols0;
	int idx_end1 = col1 + (end_row) * n_cols1;
	int value0;

	if (swap_dir == 1){
		for (int k=shift_start_offset; k < 0; k++){
			value0 = array0[idx_end0 + (k+swap_dir) * n_cols0];
			if (value0 > 0){
	
				if (bprint){
					printf("\nN_rep[%d, %d] = %d -> %d, G_swap_index[%d, %d] = %d -> %d",
						end_row + k, idx_end0, array0[idx_end0 + k * n_cols0], value0,
						end_row + k, idx_end1, array1[idx_end1 + k * n_cols1], array1[idx_end1 + (k+swap_dir) * n_cols1]
					);
		
				}
	
				array0[idx_end0 + k * n_cols0] = value0;
				array1[idx_end1 + k * n_cols1] = array1[idx_end1 + (k+swap_dir) * n_cols1];
			}
		
		}
	}
	
	else if (swap_dir == -1){
		for (int k=0; k > shift_start_offset; k--){
			value0 = array0[idx_end0 + (k+swap_dir) * n_cols0];
			if (value0 > 0){
	
				if (bprint){
					printf("\nN_rep[%d, %d] = %d -> %d, G_swap_index[%d, %d] = %d -> %d",
						end_row + k, idx_end0, array0[idx_end0 + k * n_cols0], value0,
						end_row + k, idx_end1, array1[idx_end1 + k * n_cols1], array1[idx_end1 + (k+swap_dir) * n_cols1]
					);
		
				}
	
				array0[idx_end0 + k * n_cols0] = value0;
				array1[idx_end1 + k * n_cols1] = array1[idx_end1 + (k+swap_dir) * n_cols1];
			}
		
		}
	}

}


__device__ void generate_synapses(
	const int N,
	const int n,
	const int neuron_idx,
	int* N_rep,
	int* G_swap_tensor,
	int& swap_src_N_s_start, int& swap_snk_N_s_start,
	int& swap_src_G_count, int& swap_snk_G_count,
	const int max_snk_count,
	hiprandState &local_state,
	int G_swap_tensor_shape_1, 
	const int swap_type,
	const int index_offset,
	const int relative_index_offset,
	const int swap_dir,
	bool bprint
){
	
	int snk_N;
	int min_G_swap_snk = G_swap_tensor[neuron_idx + swap_snk_N_s_start * G_swap_tensor_shape_1];
	int max_G_swap_snk = G_swap_tensor[neuron_idx + (swap_snk_N_s_start + swap_snk_G_count - 1) * G_swap_tensor_shape_1];
	if (swap_snk_G_count == 0){
		min_G_swap_snk = max_snk_count + relative_index_offset;
		max_G_swap_snk = -1;
	}
	float r;

	int s_end = swap_src_N_s_start + swap_src_G_count;
	

	for (int s=swap_src_N_s_start; s < s_end; s++){
	// for (int s=swap_src_N_s_start; s < swap_src_N_s_start + 2; s++){
		
		r = hiprand_uniform(&local_state);

		snk_N = __float2int_rd(r * __int2float_rn(max_snk_count)) + relative_index_offset;
			
				
		if (bprint) printf("\n[%d, %d] new=%d (%f), t=%d, s=%d, [%d, %d], [offset = %d - %d]", 
						   n, neuron_idx, snk_N, r, swap_type, s,
						   min_G_swap_snk, max_G_swap_snk, 
						   index_offset, relative_index_offset);

		if (swap_snk_G_count < max_snk_count)
		{	
			bool found = false;
			int i = 0;	
			int j = 0;
			int swap_idx = neuron_idx + (swap_snk_N_s_start)  * G_swap_tensor_shape_1;
			int G_swap0;
			int G_swap_m1;

			int write_row;

			int last_write_mode = 0;
			int write_mode = 0;

			// while ((!found) && (j < 40)){
			while ((!found) && (j < G_swap_tensor_shape_1)){
				
				write_mode = 0;
				//write_row = s - s_offset;
				// write = -i;
				swap_idx = neuron_idx + (swap_snk_N_s_start + i )  * G_swap_tensor_shape_1;
				
				G_swap0 = G_swap_tensor[swap_idx];
				G_swap_m1 = G_swap_tensor[swap_idx - G_swap_tensor_shape_1];


				if((snk_N < min_G_swap_snk) || (swap_snk_G_count == 0)){
				
	
					min_G_swap_snk = snk_N;
					
					if (swap_dir == 1){
						write_row = swap_snk_N_s_start - 1;
					} else {
						write_row = swap_snk_N_s_start;
					}
					write_mode = 1;

					if (swap_snk_G_count == 0){
						max_G_swap_snk = snk_N;
					}
					// G_swap_tensor[swap_idx - G_swap_tensor_shape_1] = G_swap0;	
					// G_swap_tensor[swap_idx] = snk_N;		
				}
				else if((snk_N > max_G_swap_snk)){
					write_mode = 2;
					// if (swap_snk_G_count == 0){
					// 	min_G_swap_snk = snk_N;
					// }
					max_G_swap_snk = snk_N;
					if (swap_dir == 1){
						write_row = swap_snk_N_s_start + swap_snk_G_count - 1;
					} else {
						write_row = swap_snk_N_s_start + swap_snk_G_count;
					}
					

				}
				else if ((G_swap_m1 < snk_N) && (snk_N < G_swap0)){
					write_mode = 3;
					if (swap_dir == 1){
						write_row = swap_snk_N_s_start + i - 1;
					} else {
						write_row = swap_snk_N_s_start + i;
					}
					
				}

				found = write_mode > 0;

				if (found){
					if (swap_dir == 1){
						swap_snk_N_s_start -= 1;
					} else {
						swap_src_N_s_start += 1;
					}
					
					// write = snk_N;
					// s_offset++;
					swap_snk_G_count++;
					swap_src_G_count--;
					// G_swap_tensor[neuron_idx + (write_row) * G_swap_tensor_shape_1] = write;
					break;}
				

				if ((snk_N == G_swap0)){
					snk_N = (snk_N + 1) % max_snk_count;
				}

				// if (bprint || (j >= 30)) {
				if (bprint) {
					printf("\n[%d, %d] + new=%d[i=%d, write_mode=%d] G_swap_m1=%d, G_swap0=%d, [%d, %d], max_snk_count=%d, (%d), swap_snk_G_count=%d, s=%d", 
						n, neuron_idx, snk_N, i, write_mode, G_swap_m1, G_swap0, 
						min_G_swap_snk,
						max_G_swap_snk,
						max_snk_count, swap_type, swap_snk_G_count, s);
				}
				
				i = (i + 1) % swap_snk_G_count;
				j++;

				// if (j >= 10){
				// 	printf("\nn=%d; new=%d[%d] G_swap0=%d, max_snk_count=%d, (%d), s=%d", 
				// 		   n, snk_N, i, G_swap0, max_snk_count, swap_type, s);
				// }
			}

			// if (bprint || (j >= 30)) {
			if (false) {
				printf("\n[%d, %d] (found j=%d, mod:%d->%d) N_rep[%d, %d]=%d (%d) [%d (snk_N) + %d - %d]", 
					n, neuron_idx, j, last_write_mode, write_mode,
					write_row, n, N_rep[n + (write_row) * N], N_rep[n + (swap_snk_N_s_start-1) * N], snk_N,
					index_offset, relative_index_offset);
			}

			//|| (j >= 30)
			if ((swap_dir > 0) && (write_mode > 1)){
				shift_values_row_wise_(
					swap_snk_N_s_start - write_row - 1,
					N_rep, G_swap_tensor,
					n, neuron_idx,
					N, G_swap_tensor_shape_1,
					write_row,
					swap_dir,
					bprint
				);
			} else if ((swap_dir < 0) && (write_mode > 0) && (write_mode != 2)){
				shift_values_row_wise_(
					write_row - swap_src_N_s_start - 1,
					N_rep, G_swap_tensor,
					n, neuron_idx,
					N, G_swap_tensor_shape_1,
					swap_src_N_s_start,
					swap_dir,
					bprint
				);
			}


			N_rep[n + (write_row) * N] = snk_N + index_offset - relative_index_offset;
			G_swap_tensor[neuron_idx + (write_row) * G_swap_tensor_shape_1] = snk_N;

			// bprint || (j >= 30)

			if (bprint  ) {
				printf("\n[%d, %d] (found j=%d, mod:%d->%d) N_rep[%d, %d]=%d (%d) [%d (snk_N) + %d - %d]", 
					n, neuron_idx, j, last_write_mode, write_mode,
					write_row, n, N_rep[n + (write_row) * N], N_rep[n + (swap_snk_N_s_start-1) * N], snk_N,
					index_offset, relative_index_offset);
			}
			last_write_mode = write_mode;
		} 

		// 	swap_snk_G_count++;
	}

}



__global__ void swap_groups_(
	const long* neurons, const int n_neurons, 
	const long* groups, const int n_groups,
	const int* neuron_group_indices,
	int* G_swap_tensor, const int G_swap_tensor_shape_1,
	const float* swap_rates,
	const int* group_neuron_counts_inh, const int* group_neuron_counts_exc, const int* group_neuron_counts_total, 
	const int* G_delay_distance,
	const int* N_relative_G_indices, const int* G_neuron_typed_ccount,
	int N,
	int G,
	int S,
	int D,
	int* N_flags,
	int* N_rep,
	int* N_delays,
	hiprandState* randstates,
	int* neuron_group_counts,
	const int expected_snk_type,
	const int print_idx,
	const int N_flags_row_type = 1,
	const int N_flags_row_group = 2
){
	const int neuron_idx = blockIdx.x * blockDim.x + threadIdx.x; 

	if (neuron_idx < n_neurons){

		// bool bprint = (neuron_idx == min(print_idx, n_neurons- 1));
		bool bprint = false;

		const int n = neurons[neuron_idx];
		
		const int group_index = neuron_group_indices[neuron_idx];
		const int snk_group_index = group_index + 2 * n_groups;

		const int swap_src_G = groups[group_index];
		const int src_G = groups[snk_group_index - n_groups];
		const int swap_snk_G = groups[snk_group_index];

		const float swap_rate = swap_rates[group_index];

		const int total_src_G_count = group_neuron_counts_total[group_index];
		const int total_snk_G_count = group_neuron_counts_total[snk_group_index];

		if (bprint){		
			printf("\n\nswap_src %d (%d), src_G %d %d (%d), swap_snk %d (%d)  neuron_group_indices[%d] = %d\n", 
			swap_src_G, total_src_G_count,
			N_flags[n + N_flags_row_group * N], src_G, group_neuron_counts_total[snk_group_index - n_groups],
			swap_snk_G, total_snk_G_count, neuron_idx, (int)neuron_group_indices[neuron_idx]);
		}

		int snk_N;
		int snk_type;
		int snk_G;

		int swap_delay_src = G_delay_distance[swap_src_G + src_G * G];
		int swap_delay_snk = G_delay_distance[swap_snk_G + src_G * G];

		int s_start = N_delays[n + min(swap_delay_src, swap_delay_snk) * N];
		int s_end =  N_delays[n + (max(swap_delay_src, swap_delay_snk) + 1) * N];

		int swap_src_N_s_start = s_start;
		int swap_snk_N_s_start = s_start;

		int swap_src_G_count = 0;
		int swap_snk_G_count = 0;

		for (int s=s_start; s < s_end; s++)
		{
			
			snk_N = N_rep[n + s * N];
			snk_type = N_flags[snk_N * 2];
			

			if (snk_type == expected_snk_type){
				
				
				snk_G = N_flags[snk_N + N_flags_row_group * N];

				if (snk_G == swap_src_G)
				{
					
					if (swap_src_G_count == 0){
						swap_src_N_s_start = s;
					}
					swap_src_G_count += 1;

					G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] = -total_src_G_count-N_rep[n + s * N]; //-2;
					if (bprint) printf("\n(%d) n_snk=%d, (snk_G=%d)  (s=%d) %d %d, src_counts=[%d, ]", 
						n, N_rep[n + s * N], snk_G, s, snk_G == swap_src_G, snk_G == swap_snk_G, 
						swap_src_G_count);
					
					N_rep[n + s * N] = -1;
					
				}
				else if (snk_G == swap_snk_G)
				{
					if (snk_type == expected_snk_type){
						if (swap_snk_G_count == 0){
							swap_snk_N_s_start = s;
						}
						swap_snk_G_count += 1;
					} 

					G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] = N_relative_G_indices[snk_N];
					
					if (bprint) printf("\n(%d) n_snk=%d, (snk_G=%d)  (s=%d) %d %d, snk_N_rel=%d", 
						n, N_rep[n + s * N], snk_G, s, 
						snk_G == swap_src_G, snk_G == swap_snk_G, 
						N_relative_G_indices[snk_N]);
				} 
				else if((swap_src_G_count > 0) || (swap_snk_G_count > 0)){
					G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] = N_rep[n + s * N];	
				}
			} 
			else if((swap_src_G_count > 0) || (swap_snk_G_count > 0))
			{
				G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] = N_rep[n + s * N];	
			}
			
		}

		if (swap_snk_G_count == 0){
			swap_snk_N_s_start = swap_src_N_s_start + swap_src_G_count;
		}
		// if (swap_snk_G_count_exc == 0){
		// 	swap_snk_N_s_start_exc += 1;
		// }

		if (swap_rate < 1.f){
			s_end = swap_src_N_s_start + swap_src_G_count;

			swap_src_G_count = __float2int_rd (__int2float_rn(swap_src_G_count) * swap_rate);

			for (int s=swap_src_N_s_start + swap_src_G_count; s < s_end; s++)
			{
				snk_N = - G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] - total_src_G_count;
				G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] = snk_N;
				N_rep[n + s * N] =  snk_N;
			}
		}


		if (bprint){
			printf("\n\nrow intervals: src=[%d, %d (+%d)) snk=[%d, %d (+%d)), swap_rate=%f\n", 
				   swap_src_N_s_start, swap_src_N_s_start + swap_src_G_count, 
				   swap_src_G_count, 
				   swap_snk_N_s_start, swap_snk_N_s_start + swap_snk_G_count,
				   swap_snk_G_count, swap_rate);
					//    printf("exc: src=[%d, +%d] snk=[%d, +%d]\n", 
					//    swap_src_N_s_start_exc, swap_src_G_count_exc, swap_snk_N_s_start_exc, swap_snk_G_count_exc);
		}

		if (swap_src_G_count > 0){

			int distance = max(swap_snk_N_s_start - (swap_src_N_s_start + swap_src_G_count), 
						       min(0, swap_snk_N_s_start + swap_snk_G_count - swap_src_N_s_start));

			int swap_dir = 1 * (swap_snk_N_s_start > swap_src_N_s_start) + -1 * (swap_snk_N_s_start < swap_src_N_s_start);
			
			if (distance != 0){

				// if (swap_dir == 1){
					roll_copy(
						N_rep, G_swap_tensor, 
						n, neuron_idx, 
						min(swap_src_N_s_start, swap_snk_N_s_start + swap_snk_G_count), 
						N, G_swap_tensor_shape_1, 
						(swap_snk_N_s_start - swap_src_N_s_start) * (swap_dir == 1) + (swap_dir == -1) * (-distance + swap_src_G_count), 
						swap_src_G_count * (swap_dir == 1) - distance *  (swap_dir == -1), 
						bprint);
				// } else {
				// 	roll_copy(
				// 		N_rep, G_swap_tensor, 
				// 		n, neuron_idx, 
				// 		swap_snk_N_s_start, 
				// 		N, G_swap_tensor_shape_1, 
				// 		swap_snk_N_s_start - swap_src_N_s_start, 
				// 		swap_dir * swap_src_G_count, 
				// 		bprint);
				// }


				swap_src_N_s_start += distance;

				if (bprint) {printf("\n\nswap_src_N_s_start=%d, distance=%d\n", swap_src_N_s_start, distance);}

			}


			hiprandState local_state = randstates[neuron_idx];

			int max_snk_count;
			int index_offset;
			int relative_index_offset;
			if (expected_snk_type == 1){
				max_snk_count = group_neuron_counts_inh[snk_group_index];
				index_offset = G_neuron_typed_ccount[swap_snk_G];
				relative_index_offset = 0;
			}
			else if (expected_snk_type == 2){
				max_snk_count = group_neuron_counts_exc[snk_group_index];
				index_offset = G_neuron_typed_ccount[G + swap_snk_G];
				relative_index_offset = group_neuron_counts_inh[snk_group_index];
			}

			if (swap_src_G_count > 0){
				generate_synapses(
					N, n,
					neuron_idx, N_rep,
					G_swap_tensor,
					swap_src_N_s_start, swap_snk_N_s_start,
					swap_src_G_count, swap_snk_G_count,
					max_snk_count,
					local_state,
					G_swap_tensor_shape_1,
					expected_snk_type,
					index_offset,
					relative_index_offset,
					swap_dir,
					bprint
				);
			}

			randstates[neuron_idx] = local_state;
		}

		bool count = true;

		if (count){

			int swap_src_G_count = 0;
			int swap_snk_G_count = 0;
		
			for (int s=0; s < S; s++){
				snk_G = N_flags[N_rep[n + s * N] + N_flags_row_group * N]; 
				G_swap_tensor[neuron_idx + s * G_swap_tensor_shape_1] = snk_G;
				swap_src_G_count += (snk_G == swap_src_G);
				swap_snk_G_count += (snk_G == swap_snk_G);
			}	
			neuron_group_counts[neuron_idx] = swap_src_G_count;
			neuron_group_counts[neuron_idx + G_swap_tensor_shape_1] = swap_snk_G_count;

			if (swap_delay_src != swap_delay_snk){
				
				int d1 = max(swap_delay_src, swap_delay_snk);
				int count0 = swap_delay_src;

				for (int d=min(swap_delay_src, swap_delay_snk); d < d1; d++){
					N_delays[n + d * N] -= count0;
				}
			}
		}
	
	}
}


void SnnRepresentation::swap_groups(
	long* neurons, const int n_neurons, 
	long* groups, const int n_groups, 
	int* neuron_group_indices,
	int* G_swap_tensor, const int G_swap_tensor_shape_1,
	float* swap_rates_inh, float* swap_rates_exc,
	int* group_neuron_counts_inh, int* group_neuron_counts_exc, int* group_neuron_counts_total,
	int* G_delay_distance,
	int* N_relative_G_indices, int* G_neuron_typed_ccount,
	int* neuron_group_counts,
	const int print_idx
)
{
	LaunchParameters lp_swap_groups = LaunchParameters(n_neurons, (void *)swap_groups_);

	//printf("\nswap groups %d, %d\n", n_groups, n_neurons);

	swap_groups_ KERNEL_ARGS2(lp_swap_groups.grid3, lp_swap_groups.block3)(
		neurons, n_neurons,
		groups, n_groups,
		neuron_group_indices,
		G_swap_tensor, G_swap_tensor_shape_1,
		swap_rates_inh,
		group_neuron_counts_inh, group_neuron_counts_exc, group_neuron_counts_total,
		G_delay_distance,
		N_relative_G_indices, G_neuron_typed_ccount,
		N,
		G,
		S,
		D,
		N_flags,
		N_rep,
		N_delays,
		rand_states,
		neuron_group_counts,
		1,
		print_idx
	);

	checkCudaErrors(hipDeviceSynchronize());

	swap_groups_ KERNEL_ARGS2(lp_swap_groups.grid3, lp_swap_groups.block3)(
		neurons, n_neurons,
		groups, n_groups,
		neuron_group_indices,
		G_swap_tensor, G_swap_tensor_shape_1,
		swap_rates_exc,
		group_neuron_counts_inh, group_neuron_counts_exc, group_neuron_counts_total,
		G_delay_distance,
		N_relative_G_indices, G_neuron_typed_ccount,
		N,
		G,
		S,
		D,
		N_flags,
		N_rep,
		N_delays,
		rand_states,
		neuron_group_counts,
		2,
		print_idx
	);

	checkCudaErrors(hipDeviceSynchronize());
}

void SnnRepresentation::swap_groups_python(
	long neurons, const int n_neurons, 
	long groups, const int n_groups, 
	const long neuron_group_indices,
	const long G_swap_tensor, const int G_swap_tensor_shape_1,
	const long swap_rates_inh, const long swap_rates_exc,
	const long group_neuron_counts_inh, const long group_neuron_counts_exc, const long group_neuron_counts_total,
	const long G_delay_distance, 
	const long N_relative_G_indices, const long G_neuron_typed_ccount,
	long neuron_group_counts,
	const int print_idx
)
{
	swap_groups(reinterpret_cast<long*> (neurons), n_neurons, 
				reinterpret_cast<long*> (groups), n_groups, 
				reinterpret_cast<int*> (neuron_group_indices),
				reinterpret_cast<int*> (G_swap_tensor), G_swap_tensor_shape_1,
				reinterpret_cast<float*> (swap_rates_inh), reinterpret_cast<float*> (swap_rates_exc),
				reinterpret_cast<int*> (group_neuron_counts_inh), reinterpret_cast<int*> (group_neuron_counts_exc), reinterpret_cast<int*> (group_neuron_counts_total),
				reinterpret_cast<int*> (G_delay_distance),
				reinterpret_cast<int*> (N_relative_G_indices), reinterpret_cast<int*> (G_neuron_typed_ccount),
				reinterpret_cast<int*> (neuron_group_counts),
				print_idx
				
	);
}


__global__ void reset_N_rep_pre_synaptic_arrays(
	const int N,
	const int S,
	int* Buffer,
	int* N_rep_pre_synaptic,
	int* N_rep_pre_synaptic_idcs,
	int* N_rep_pre_synaptic_counts
){
	const int src_N = blockIdx.x * blockDim.x + threadIdx.x; 
	if (src_N < N){
		for (int s = 0; s < S; s++){
			Buffer[src_N + s * N] = -1;
			N_rep_pre_synaptic[src_N + s * N] = -1;
			N_rep_pre_synaptic_idcs[src_N + s * N] = -1;
		}

		if (src_N == 0){
			N_rep_pre_synaptic_counts[0] = 0;
		}
		N_rep_pre_synaptic_counts[src_N + 1] = 0;

	}

}


__global__ void reset_N_rep_snk_counts(
	const int N,
	int* N_rep_pre_synaptic_counts
){
	const int src_N = blockIdx.x * blockDim.x + threadIdx.x; 
	
	if (src_N < N){

		if (src_N == 0){
			N_rep_pre_synaptic_counts[0] = 0;
		}

		N_rep_pre_synaptic_counts[src_N + 1] = 0;
	}
}


__global__ void fill_N_rep_snk_counts(
	const int N,
	const int S,
	int* N_rep,
	int* N_rep_pre_synaptic_counts
){
	const int src_N = blockIdx.x * blockDim.x + threadIdx.x; 
	int snk_N;
	
	if (src_N < N){

		for (int s = 0; s < S; s++){
			snk_N = N_rep[src_N + s * N];

			if (snk_N == -1){
				printf("\n %d", src_N);
			}

			atomicAdd(&N_rep_pre_synaptic_counts[snk_N + 1], 1);
		}
	}
}


__global__ void fill_unsorted_N_rep_pre_synaptic_idcs(
	const int N,
	const int S,
	int* N_rep,
	int* SortBuffer,
	int* N_rep_pre_synaptic_idcs,
	int* N_rep_pre_synaptic_counts
){

	const int src_N = blockIdx.x * blockDim.x + threadIdx.x; 

	if (src_N < N){

		int snk_N;
		int write_idx;
	
		int synapse_idx;

		for (int s = 0; s < S; s++){
			
			synapse_idx = src_N + s * N;

			snk_N = N_rep[synapse_idx];
			write_idx = N_rep_pre_synaptic_counts[snk_N];
			
			while (synapse_idx != -1){
				
				synapse_idx = atomicExch(&N_rep_pre_synaptic_idcs[write_idx], synapse_idx);
				SortBuffer[write_idx] = snk_N;
				write_idx++;
			}

			atomicAdd(&N_rep_pre_synaptic_counts[snk_N],1);

		}
	}

}


__global__ void fill_N_rep_pre_synaptic(
	const int N,
	const int S,
	int* N_rep,
	int* N_rep_pre_synaptic,
	int* N_rep_pre_synaptic_idcs,
	int* N_rep_pre_synaptic_counts
){

	const int src_N = blockIdx.x * blockDim.x + threadIdx.x; 

	if (src_N < N){

		int snk_N;
		int write_idx;
	
		int synapse_idx;

		for (int s = 0; s < S; s++){
			
			synapse_idx = src_N + s * N;

			snk_N = N_rep[synapse_idx];
			write_idx = N_rep_pre_synaptic_counts[snk_N];
			
			while (N_rep_pre_synaptic_idcs[write_idx] != synapse_idx){
				write_idx++;
			}

			N_rep_pre_synaptic[write_idx] = src_N;

		}
	}

}


void sort_N_rep_sysnaptic(
	const int N,
	const int S,
	int* sort_keys_buffer,
	int* N_rep_pre_synaptic_idcs,
	int* N_rep_pre_synaptic_counts,
	const bool verbose = true
){

	auto sort_keys_buffer_dp = thrust::device_pointer_cast(sort_keys_buffer);
	auto N_rep_dp = thrust::device_pointer_cast(N_rep_pre_synaptic_idcs);
	auto N_rep_counts_dp = thrust::device_pointer_cast(N_rep_pre_synaptic_counts);

	int n_sorted = 0;
	int N_batch_size = 50000;
	int S_batch_size;

	std::string msg;
	if (verbose) {
		msg = "sorted: 0/" + std::to_string(N);
		std::cout << msg;
	}

	while (n_sorted < N){
			
	 	if (n_sorted + N_batch_size > N){
	 		N_batch_size = N - n_sorted;
		} 

		// printf("\nN_batch_size=%d", N_batch_size);

		S_batch_size = N_rep_counts_dp[n_sorted + N_batch_size] - N_rep_counts_dp[n_sorted];

		// printf("\nS_batch_size=%d\n", S_batch_size);

	 	thrust::stable_sort_by_key(N_rep_dp, N_rep_dp + S_batch_size, sort_keys_buffer_dp);
	 	thrust::stable_sort_by_key(sort_keys_buffer_dp, sort_keys_buffer_dp + S_batch_size, N_rep_dp);
		
	 	n_sorted += N_batch_size;
	 	sort_keys_buffer_dp += S_batch_size;
	 	N_rep_dp += S_batch_size;

	 	if (verbose) { 
	 		std::cout << std::string(msg.length(),'\b');
	 		msg = "sorted: " + std::to_string(n_sorted) + "/" + std::to_string(N);
	 		std::cout << msg;
	 	}
	}

	if (verbose) printf("\n");

}


void SnnRepresentation::actualize_N_rep_pre_synaptic(){

	LaunchParameters launch_pars = LaunchParameters(N, (void *)reset_N_rep_pre_synaptic_arrays);

	checkCudaErrors(hipDeviceSynchronize());

	reset_N_rep_pre_synaptic_arrays KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N,
		S,
		N_rep_buffer,
		N_rep_pre_synaptic,
		N_rep_pre_synaptic_idcs,
		N_rep_pre_synaptic_counts
	);

	checkCudaErrors(hipDeviceSynchronize());

	fill_N_rep_snk_counts KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N,
		S,
		N_rep,
		N_rep_pre_synaptic_counts
	);

	thrust::device_ptr<int> count_dp = thrust::device_pointer_cast(N_rep_pre_synaptic_counts);

	checkCudaErrors(hipDeviceSynchronize());

	thrust::inclusive_scan(thrust::device, count_dp, count_dp + N + 1, count_dp);

	checkCudaErrors(hipDeviceSynchronize());
	printf("\nfill (unsorted) N_rep_pre_synaptic_idcs...");

	fill_unsorted_N_rep_pre_synaptic_idcs KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N,
		S,
		N_rep,
		N_rep_buffer,
		N_rep_pre_synaptic_idcs,
		N_rep_pre_synaptic_counts
	);

	checkCudaErrors(hipDeviceSynchronize());

	reset_N_rep_snk_counts KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N,
		N_rep_pre_synaptic_counts
	);	

	checkCudaErrors(hipDeviceSynchronize());

	fill_N_rep_snk_counts KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N,
		S,
		N_rep,
		N_rep_pre_synaptic_counts
	);

	checkCudaErrors(hipDeviceSynchronize());

	thrust::inclusive_scan(thrust::device, count_dp, count_dp + N + 1, count_dp);

	checkCudaErrors(hipDeviceSynchronize());

	sort_N_rep_sysnaptic(N, S, N_rep_buffer, N_rep_pre_synaptic_idcs, N_rep_pre_synaptic_counts);

	checkCudaErrors(hipDeviceSynchronize());

	printf("\nfill N_rep_pre_synaptic...");

	fill_N_rep_pre_synaptic KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N,
		S,
		N_rep,
		N_rep_pre_synaptic,
		N_rep_pre_synaptic_idcs,
		N_rep_pre_synaptic_counts
	);

	checkCudaErrors(hipDeviceSynchronize());


	// std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	// std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	printf(" done.\n");
	//std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
}





__global__ void remove_all_synapses_to_group_(
	const int N,
	const int S,
	const int* N_flags,
	int* N_rep,
	int* N_rep_pre_synaptic,
	int* N_rep_pre_synaptic_idcs,
	int* N_rep_pre_synaptic_counts,
	const int group,
	const int N_flags_row_group = 2,
	const int delete_synapse_value = -2
){
	const int n = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (n < N)
	{
		for (int s = 0; s < S; s++){
			if (N_flags[N_rep[n  + s * N] + N_flags_row_group * N] == group){
				N_rep[n  + s * N] = -2;
			}
		}

		if (N_flags[n + N_flags_row_group * N] == group){		
			int s_end2 = N_rep_pre_synaptic_counts[n + 1];
			for (int s2 = N_rep_pre_synaptic_counts[n]; s2 < s_end2; s2++){
				N_rep_pre_synaptic[N_rep_pre_synaptic_idcs[s2]] = -2;
			}
		}
	}
}

void SnnRepresentation::remove_all_synapses_to_group(const int group){

	checkCudaErrors(hipDeviceSynchronize());

	LaunchParameters launch_pars = LaunchParameters(N, (void *)remove_all_synapses_to_group_);

	remove_all_synapses_to_group_ KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N, S,
		N_flags, 
		N_rep,
		N_rep_pre_synaptic,
		N_rep_pre_synaptic_idcs,
		N_rep_pre_synaptic_counts,
		group
	);

	checkCudaErrors(hipDeviceSynchronize());

}


__global__ void nullify_all_weights_to_group_(
	const int N,
	const int S,
	const int* N_flags,
	int* N_rep,
	float* N_weights,
	const int group,
	const int N_flags_row_group = 2,
	const int delete_synapse_value = -2
){
	const int n = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (n < N)
	{
		for (int s = 0; s < S; s++){
			if (N_flags[N_rep[n  + s * N] + N_flags_row_group * N] == group){
				N_weights[n  + s * N] = 0.f;
			}
		}

	}
}

void SnnRepresentation::nullify_all_weights_to_group(const int group){

	checkCudaErrors(hipDeviceSynchronize());

	LaunchParameters launch_pars = LaunchParameters(N, (void *)nullify_all_weights_to_group_);

	nullify_all_weights_to_group_ KERNEL_ARGS2(launch_pars.grid3, launch_pars.block3)(
		N, S,
		N_flags, 
		N_rep,
		N_weights,
		group
	);

	checkCudaErrors(hipDeviceSynchronize());

}
